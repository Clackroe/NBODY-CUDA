#include "hip/hip_runtime.h"
#include <vector>

#define USE_CUDA
#include <Body.hpp>
#include <Vec.hpp>

#include <stdio.h>

#define C_CHECK(ret)                                                                     \
    {                                                                                    \
        auto err = ret;                                                                  \
        if (err != hipSuccess) {                                                        \
            printf("Cuda err in %s:%i %s", __FILE__, __LINE__, hipGetErrorString(err)); \
        }                                                                                \
    }

__global__ void CalcForcesKernel(Body* bodies, Vec2* accels, int n, double G)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n)
        return;
    Vec2 acc = { 0, 0 };

    for (int j = 0; j < n; j++) {
        if (i == j) {
            continue;
        }
        Body b1 = bodies[i];
        Body b2 = bodies[j];
        double force = Force(b1, b2, G);
        Vec2 dir = Direction(b1.position, b2.position);
        double acc1 = force / b1.mass;
        acc = add(acc, scale(dir, acc1));
    }

    accels[i] = acc;
}

__global__ void UpdateKernel(Body* bodies, Vec2* accels, double deltaTime, int width, int height, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    Body& b = bodies[i];

    b.velocity = add(b.velocity, accels[i]);

    if (b.position.x < 0 || b.position.x > width) {
        b.velocity.x *= -1;
    }
    if (b.position.y < 0 || b.position.y > height) {
        b.velocity.y *= -1;
    }

    b.position = add(b.position, scale(b.velocity, deltaTime));
}

Body* g_DeviceBodies;
Vec2* g_DeviceAccels;

void CudaInit(int maxBodies)
{
    C_CHECK(hipMalloc(&g_DeviceBodies, maxBodies * sizeof(Body)));
    C_CHECK(hipMalloc(&g_DeviceAccels, maxBodies * sizeof(Vec2)));
}

void CudaShutdown()
{
    C_CHECK(hipFree(g_DeviceBodies));
    C_CHECK(hipFree(g_DeviceAccels));
}

double CalculateForcesCuda(std::vector<Body>& bodies, double G)
{

    hipEvent_t start, end;
    int n = bodies.size();
    C_CHECK(hipMemcpy(g_DeviceBodies, bodies.data(), n * sizeof(Body), hipMemcpyHostToDevice));
    // Don't need to set accels since they are set to zero initially during the simulation

    // Dont want to include the MemCPY in the final time

    C_CHECK(hipEventCreate(&start));
    C_CHECK(hipEventCreate(&end));

    C_CHECK(hipEventRecord(start));

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    CalcForcesKernel<<<numBlocks, blockSize>>>(g_DeviceBodies, g_DeviceAccels, n, G);

    C_CHECK(hipEventRecord(end));
    C_CHECK(hipDeviceSynchronize());

    float time;
    C_CHECK(hipEventElapsedTime(&time, start, end));
    double seconds = time / 1000;

    C_CHECK(hipEventDestroy(start));
    C_CHECK(hipEventDestroy(end));

    return seconds;
}

double UpdateCuda(std::vector<Body>& bodies, double deltaTime, int width, int height)
{

    hipEvent_t start, end;

    C_CHECK(hipEventCreate(&start));
    C_CHECK(hipEventCreate(&end));

    C_CHECK(hipEventRecord(start));

    int n = bodies.size();

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    UpdateKernel<<<numBlocks, blockSize>>>(g_DeviceBodies, g_DeviceAccels, deltaTime, width, height, n);

    C_CHECK(hipEventRecord(end));
    C_CHECK(hipDeviceSynchronize());

    float time;
    C_CHECK(hipEventElapsedTime(&time, start, end));
    double seconds = time / 1000;

    // Not including memcpy in the timings
    C_CHECK(hipMemcpy(bodies.data(), g_DeviceBodies, n * sizeof(Body), hipMemcpyDeviceToHost));

    C_CHECK(hipEventDestroy(start));
    C_CHECK(hipEventDestroy(end));

    return seconds;
}
